
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUDA
#include "oneflow/core/kernel/util/cuda_half_util.h"
#include "oneflow/core/cuda/elementwise.cuh"
#include "oneflow/user/kernels/selu_kernel.h"
namespace oneflow {

namespace {

template<template<typename> class Opt, typename T>
struct ElemwiseSeluFunctor<DeviceType::kGPU, Opt, T> final {
  void operator()(DeviceCtx* ctx, const int64_t elem_cnt, T lambda, T alpha, T* out,
                  const T* in) {
    OF_CUDA_CHECK(oneflow::cuda::elementwise::Unary(SeluFunctor<T>(lambda, alpha), elem_cnt,
                                                    out, in, ctx->cuda_stream()));
  }
};

template<template<typename> class Opt, typename T>
struct ElemwiseSeluGradFunctor<DeviceType::kGPU, Opt, T> final {
  void operator()(DeviceCtx* ctx, const int64_t elem_cnt, T lambda, T alpha, T* dx, const T* y,
                  const T* dy) {
    OF_CUDA_CHECK(oneflow::cuda::elementwise::Binary(SeluGradFunctor<T>(lambda, alpha),
                                                     elem_cnt, dx, y, dy, ctx->cuda_stream()));
  };
};

}  // namespace

REGISTER_SELU_KERNELS(DeviceType::kGPU, half);
REGISTER_SELU_KERNELS(DeviceType::kGPU, float);
REGISTER_SELU_KERNELS(DeviceType::kGPU, double);

}  // namespace oneflow
#endif  // WITH_CUDA
